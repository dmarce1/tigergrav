#include "hip/hip_runtime.h"
#include <tigergrav/options.hpp>
#include <tigergrav/cuda_export.hpp>
#include <tigergrav/cuda_check.hpp>
#include <tigergrav/gravity_cuda.hpp>
#include <tigergrav/green.hpp>

CUDA_EXPORT expansion<float> green_ewald(const vect<float> &X) {
	static const float three(3.0);
	const float fouroversqrtpi(4.0 / sqrt(M_PI));
	static const float two(2.0);
	static const float eight(8.0);
	static const float fifteen(15.0);
	static const float thirtyfive(35.0);
	static const float fourty(40.0);
	static const float fiftysix(56.0);
	static const float sixtyfour(64.0);
	static const float onehundredfive(105.0);
	static const float rcut(1.0e-6);
	const float r = abs(X);
	const float zmask = r > rcut;											// 2
	vect<int> n;
	expansion<double> D;
	D = 0.0;
	for (n[0] = -4; n[0] <= +4; n[0]++) {
		for (n[1] = -4; n[1] <= +4; n[1]++) {
			for (n[2] = -4; n[2] <= +4; n[2]++) {
				const vect<float> dx = X - vect<float>(n);				// 3
				const float r2 = dx.dot(dx);				// 5
				const float r4 = r2 * r2;					// 1
				const float r = sqrt(r2);					// 7
				if (r < 3.6) {
					const float cmask = 1.0 - (n.dot(n) > 0.0);
					const float mask = (1.0 - (1.0 - zmask) * cmask);
					const float rinv = mask / max(r, rcut);		// 36
					const float r2inv = rinv * rinv;			// 1
					const float r3inv = r2inv * rinv;			// 1
					const float r5inv = r2inv * r3inv;			// 1
					const float r7inv = r2inv * r5inv;			// 1
					const float r9inv = r2inv * r7inv;			// 1
					const float erfc0 = erfcf(two * r);			// 76
					const float exp0 = expf(-two * two * r * r);
					const float expfactor = fouroversqrtpi * r * exp0; 	// 2
					const float d0 = -erfc0 * rinv;							// 2
					const float d1 = (expfactor + erfc0) * r3inv;			// 2
					const float d2 = -fma(expfactor, fma(eight, r2, three), three * erfc0) * r5inv;		// 5
					const float d3 = fma(expfactor, (fifteen + fma(fourty, r2, sixtyfour * r4)), fifteen * erfc0) * r7inv;		// 6
					const float d4 = -fma(expfactor, fma(eight * r2, (thirtyfive + fma(fiftysix, r2, sixtyfour * r4)), onehundredfive), onehundredfive * erfc0)
							* r9inv;		// 9
					green_deriv_ewald(D, d0, d1, d2, d3, d4, dx);			// 576
				}
			}
		}
	}
	static const float twopi = 2.0 * M_PI;
	for (n[0] = -3; n[0] <= 3; n[0]++) {
		for (n[1] = -3; n[1] <= 3; n[1]++) {
			for (n[2] = -3; n[2] <= 3; n[2]++) {
				if (n.dot(n) < 10) {
					vect<float> h = n;
					const float h2 = h.dot(h);
					const float hdotx = h.dot(X);
					if (h2 > 0.0) {
						const float co = cosf(twopi * hdotx);
						const float so = sinf(twopi * hdotx);
						float c0 = (-1.0 / M_PI) * expf(-M_PI * M_PI / 4.0 * h2) / h2;
						D() += c0 * co;
						for (int a = 0; a < NDIM; a++) {
							const float c1 = -twopi * c0 * h[a];
							D(a) += c1 * so;
							for (int b = 0; b <= a; b++) {
								const float c2 = +twopi * c1 * h[b];
								D(a, b) += c2 * co;
								for (int c = 0; c <= b; c++) {
									const float c3 = -twopi * c2 * h[c];
									D(a, b, c) += c3 * so;
									for (int d = 0; d <= c; d++) {
										const float c4 = +twopi * c3 * h[d];
										D(a, b, c, d) += c4 * co;
									}
								}
							}
						}
					}
				}
			}
		}
	}

	expansion<float> rcD;
//	if (r > rcut) {
	for (int i = 0; i < LP; i++) {
		rcD[i] = D[i];																	// 70
	}
	const auto D1 = green_direct(X);													// 167
	const float rinv = -D1();														// 2
	rcD() = (M_PI / 4.0) + rcD() + zmask * rinv;												// 2
	for (int a = 0; a < NDIM; a++) {
		rcD(a) = (rcD(a) - zmask * D1(a));												// 6
		for (int b = 0; b <= a; b++) {
			rcD(a, b) = (rcD(a, b) - zmask * D1(a, b));									// 12
			for (int c = 0; c <= b; c++) {
				rcD(a, b, c) = (rcD(a, b, c) - zmask * D1(a, b, c));					// 20
				for (int d = 0; d <= c; d++) {
					rcD(a, b, c, d) = (rcD(a, b, c, d) - zmask * D1(a, b, c, d));		// 30
				}
			}
		}
	}
//	} else {
//		for (int i = 0; i < LP; i++) {
//			rcD[i] = 0.0;																	// 70
//		}
//
//	}

	return rcD;

}

#include <tigergrav/interactions.hpp>

#include <stack>
#include <atomic>
void yield_to_hpx();

static vect<pos_type> *y_vect;
static part_iter y_begin;
static part_iter y_end;
static bool first_call = true;
static std::atomic<int> thread_cnt(0);

bool cuda_thread_count() {
	return thread_cnt;
}

void cuda_copy_particle_image(part_iter part_begin, part_iter part_end, const std::vector<vect<pos_type>> &parts) {
	y_begin = part_begin;
	y_end = part_end;
	const auto size = part_end - part_begin;
	if (first_call) {
		CUDA_CHECK(hipMalloc((void** ) &y_vect, sizeof(vect<pos_type> ) * size));
		first_call = false;
	}
	CUDA_CHECK(hipMemcpy(y_vect, parts.data(), size * sizeof(vect<pos_type> ), hipMemcpyHostToDevice));
}

#define EWALD_MAX_TBSIZE 64

#define WORKSIZE 128
#define NODESIZE 64
#define NWARP (WORKSIZE/WARPSIZE)
#define WARPSIZE 32

__global__ void CC_ewald_kernel(expansion<double> *lptr, const vect<pos_type> X, const multi_src *y, int ysize) {
	int l = threadIdx.x;
	int tb_size = blockDim.x;
	auto &L = *lptr;

	__shared__ double value[EWALD_MAX_TBSIZE];
//	double *value = reinterpret_cast<double*>(shmem);
	expansion<double> Lacc;
	const int ymax = ((ysize - 1) / tb_size + 1) * tb_size;
	for (int yi = l; yi < ymax; yi += tb_size) {
		for (int i = 0; i < LP; i++) {
			Lacc[i] = 0.0;
		}
		if (yi < ysize) {
			vect<pos_type> Y = y[yi].x;
			multipole<float> M = y[yi].m;
			vect<float> dX;
			for (int dim = 0; dim < NDIM; dim++) {
				dX[dim] = float(X[dim] - Y[dim]) * float(POS_INV); // 18
			}
			multipole_interaction(Lacc, M, dX, true);											// 251936
		}
		for (int i = 0; i < LP; i++) {
			value[l] = Lacc[i];
			__syncthreads();
			for (int N = tb_size / 2; N > 0; N >>= 1) {
				if (l < N) {
					value[l] += value[l + N];
				}
				__syncthreads();
			}
			if (l == 0) {
				L[i] += value[0];
			}
		}
	}
}

struct cuda_context_ewald {
	int ysize;
	hipStream_t stream;
	expansion<double> *L;
	multi_src *y;
	expansion<double> *Lp;
	multi_src *yp;
	cuda_context_ewald(int ys) {
		ysize = 1;
		while (ysize < ys) {
			ysize *= 2;
		}
		CUDA_CHECK(hipMalloc(&L, sizeof(expansion<double> )));
		CUDA_CHECK(hipMalloc(&y, sizeof(multi_src) * ysize));
		CUDA_CHECK(hipHostMalloc(&Lp, sizeof(expansion<double> )));
		CUDA_CHECK(hipHostMalloc(&yp, sizeof(multi_src) * ysize));
		CUDA_CHECK(hipStreamCreate(&stream));
	}
	void resize(int ys) {
		if (ys > ysize) {
			while (ysize < ys) {
				ysize *= 2;
			}
			CUDA_CHECK(hipFree(y));
			CUDA_CHECK(hipMalloc(&y, sizeof(multi_src) * ysize));
			CUDA_CHECK(hipHostFree(yp));
			CUDA_CHECK(hipHostMalloc(&yp, sizeof(multi_src) * ysize));
		}
	}
};

static std::atomic<int> lock_ewald(0);
static std::stack<cuda_context_ewald> stack_ewald;

cuda_context_ewald pop_context_ewald(int ys) {
	while (lock_ewald++ != 0) {
		lock_ewald--;
	}
	if (stack_ewald.empty()) {
		lock_ewald--;
		return cuda_context_ewald(ys);
	} else {
		auto ctx = stack_ewald.top();
		stack_ewald.pop();
		lock_ewald--;
		ctx.resize(ys);
		return ctx;
	}
}

void push_context_ewald(cuda_context_ewald ctx) {
	while (lock_ewald++ != 0) {
		lock_ewald--;
	}
	stack_ewald.push(ctx);
	lock_ewald--;
}

std::uint64_t gravity_CC_ewald_cuda(expansion<double> &L, const vect<pos_type> &x, std::vector<const multi_src*> &y) {
	auto ctx = pop_context_ewald(y.size());
	int k = 0;
	for (int i = 0; i < y.size(); i++) {
		ctx.yp[k++] = *y[i];
	}
	*ctx.Lp = L;
	CUDA_CHECK(hipMemcpyAsync(ctx.y, ctx.yp, sizeof(multi_src) * y.size(), hipMemcpyHostToDevice, ctx.stream));
	CUDA_CHECK(hipMemcpyAsync(ctx.L, ctx.Lp, sizeof(expansion<double> ), hipMemcpyHostToDevice, ctx.stream));

	const int tb_max = EWALD_MAX_TBSIZE;
	int tb_size;
	if (y.size() <= tb_max) {
		tb_size = (((y.size() - 1) / WARPSIZE) + 1) * WARPSIZE;
	} else {
		int nperthread = (y.size() - 1) / tb_max + 1;
		tb_size = (y.size() - 1) / nperthread + 1;
		tb_size = (((tb_size - 1) / WARPSIZE) + 1) * WARPSIZE;
	}
	if( tb_size > EWALD_MAX_TBSIZE) {
		printf( "Error ewald\n");
		abort();
	}

CC_ewald_kernel<<<dim3(1,1,1),dim3(tb_size,1,1),0,ctx.stream>>>(ctx.L, x, ctx.y, y.size());

																				CUDA_CHECK(hipMemcpyAsync(ctx.Lp, ctx.L, sizeof(expansion<double> ), hipMemcpyDeviceToHost, ctx.stream));

	while (hipStreamQuery(ctx.stream) != hipSuccess) {
		yield_to_hpx();
	}
	L = *ctx.Lp;
	push_context_ewald(std::move(ctx));
}

__global__ void PPPC_direct_kernel(force *F, const vect<pos_type> *x, const vect<pos_type> *y, const std::pair<part_iter, part_iter> *yiters,
		const multi_src *z, int *xindex, int *yindex, int *zindex, float m, float h, bool ewald) {
//	printf("sizeof(force) = %li\n", sizeof(force));

	const int iwarp = threadIdx.y;
	const int ui = blockIdx.x;
	const int l = iwarp * blockDim.x + threadIdx.x;
	const int n = threadIdx.x;

	__shared__ vect<pos_type>
	X[NODESIZE];
	__shared__ force
	G[NWARP][WARPSIZE];

	const auto yb = yindex[ui];
	const auto ye = yindex[ui + 1];
	const auto xb = xindex[ui];
	const auto xe = xindex[ui + 1];
	const auto xsize = xe - xb;
	{
		const float Hinv = 1.0 / h;
		const float H3inv = Hinv * Hinv * Hinv;
		const auto ymax = ((ye - yb - 1) / WORKSIZE + 1) * WORKSIZE + yb;
		if (l < xsize) {
			X[l] = x[xb + l];
		}
		__syncthreads();
		for (int yi = yb + l; yi < ymax; yi += WORKSIZE) {
			int jb, je;
			if (yi < ye) {
				jb = yiters[yi].first;
				je = yiters[yi].second;
//			memcpy(Ymem[l], y + jb, (je - jb) * sizeof(vect<pos_type> ));
			}
			for (int i = xb; i < xe; i++) {
				G[iwarp][n].phi = 0.0;
				G[iwarp][n].g = vect<float>(0.0);
				if (yi < ye) {
					for (int j = jb; j < je; j++) {
						const vect<pos_type> Y = y[j];
						vect<float> dX;
						if (ewald) {
							for (int dim = 0; dim < NDIM; dim++) {
								dX[dim] = float(X[i - xb][dim] - Y[dim]) * float(POS_INV);
							}
						} else {
							for (int dim = 0; dim < NDIM; dim++) {
								dX[dim] = (float(X[i - xb][dim]) - float(Y[dim])) * float(POS_INV);  // 15
							}
						}
						const float r2 = dX.dot(dX);								   // 5
						const float r = sqrt(r2);									   // 1
						const float rinv = float(1) / max(r, 0.5 * h);             	   // 2
						const float rinv3 = rinv * rinv * rinv;                        // 2
						float f, p;
						if (r > h) {
							f = rinv3;
							p = rinv;
						} else if (r > 0.5 * h) {
							const float roh = min(r * Hinv, 1.0);                           // 2
							const float roh2 = roh * roh;                                 // 1
							const float roh3 = roh2 * roh;                                // 1
							f = float(-32.0 / 3.0);
							f = f * roh + float(+192.0 / 5.0);						// 1
							f = f * roh + float(-48.0);								// 1
							f = f * roh + float(+64.0 / 3.0);						// 1
							f = f * roh3 + float(-1.0 / 15.0);						// 1
							f *= rinv3;														// 1
							p = float(+32.0 / 15.0);						// 1
							p = p * roh, float(-48.0 / 5.0);					// 1
							p = p * roh, float(+16.0);							// 1
							p = p * roh, float(-32.0 / 3.0);					// 1
							p = p * roh2, float(+16.0 / 5.0);					// 1
							p = p * roh, float(-1.0 / 15.0);					// 1
							p *= rinv;                                                    	// 1
						} else {
							const float roh = min(r * Hinv, 1.0);                           // 2
							const float roh2 = roh * roh;                                 // 1
							f = float(+32.0);
							f = f * roh + float(-192.0 / 5.0);						// 1
							f = f * roh2 + float(+32.0 / 3.0);						// 1
							f *= H3inv;                                                       	// 1
							p = float(-32.0 / 5.0);
							p = p * roh, float(+48.0 / 5.0);					// 1
							p = p * roh2, float(-16.0 / 3.0);					// 1
							p = p * roh2, float(+14.0 / 5.0);					// 1
							p *= Hinv;														// 1
						}
						const auto dXM = dX * m;								// 3
						for (int dim = 0; dim < NDIM; dim++) {
							G[iwarp][n].g[dim] -= dXM[dim] * f;    				// 6
						}
						// 13S + 2D = 15
						G[iwarp][n].phi -= p * m;    						// 2
					}
				}
				for (int N = WARPSIZE / 2; N > 0; N >>= 1) {
					if (n < N) {
						G[iwarp][n].g += G[iwarp][n + N].g;
						G[iwarp][n].phi += G[iwarp][n + N].phi;
					}
				}
				__syncthreads();
				for (int N = NWARP / 2; N > 0; N >>= 1) {
					if (l < N) {
						G[l][0].g += G[l + N][0].g;
						G[l][0].phi += G[l + N][0].phi;
					}
					__syncthreads();
				}
				if (l == 0) {
					for (int dim = 0; dim < NDIM; dim++) {
						F[i].g[dim] += G[0][0].g[dim];
					}
					F[i].phi += G[0][0].phi;
				}
				__syncthreads();
			}
		}
	}
	{
		const int zmax = ((zindex[ui + 1] - 1) / WORKSIZE + 1) * WORKSIZE;
		for (int zi = zindex[ui] + l; zi < zmax; zi += WORKSIZE) {
			for (int i = xb; i < xe; i++) {
				G[iwarp][n].phi = 0.0;
				G[iwarp][n].g = vect<float>(0.0);
				if (zi < zindex[ui + 1]) {
					const multipole<float> &M = z[zi].m;
					const vect<pos_type> &Y = z[zi].x;
					vect<float> dX;
					if (ewald) {
						for (int dim = 0; dim < NDIM; dim++) {
							dX[dim] = float(X[i - xb][dim] - Y[dim]) * float(POS_INV); // 18
						}
					} else {
						for (int dim = 0; dim < NDIM; dim++) {
							dX[dim] = float(X[i - xb][dim]) * float(POS_INV) - float(Y[dim]) * float(POS_INV);
						}
					}

					vect<double> g;
					double phi;
					multipole_interaction(g, phi, M, dX); // 516
					G[iwarp][n].g += g;  // 0 / 3
					G[iwarp][n].phi += phi;		          // 0 / 1
				}
				__syncthreads();
				for (int N = NWARP / 2; N > 0; N >>= 1) {
					if (l < N) {
						G[l][0].g += G[l + N][0].g;
						G[l][0].phi += G[l + N][0].phi;
					}
					__syncthreads();
				}
				if (l == 0) {
					for (int dim = 0; dim < NDIM; dim++) {
						F[i].g[dim] += G[0][0].g[dim];
					}
					F[i].phi += G[0][0].phi;
				}
				__syncthreads();
			}
		}
	}
}

struct cuda_context {
	int xsize, ysize, zsize, isize;
	hipStream_t stream;
	force *f;
	vect<pos_type> *x;
	std::pair<part_iter, part_iter> *y;
	multi_src *z;
	int *xi;
	int *yi;
	int *zi;
	force *fp;
	vect<pos_type> *xp;
	multi_src *zp;
	std::pair<part_iter, part_iter> *yp;
	int *xip;
	int *yip;
	int *zip;
	cuda_context(int xs, int ys, int zs, int is) {
		xsize = 1;
		ysize = 1;
		zsize = 1;
		isize = 1;
		while (xsize < xs) {
			xsize *= 2;
		}
		while (zsize < zs) {
			zsize *= 2;
		}
		while (ysize < ys) {
			ysize *= 2;
		}
		while (isize < is) {
			isize *= 2;
		}
		CUDA_CHECK(hipMalloc(&f, sizeof(force) * xsize));
		CUDA_CHECK(hipMalloc(&x, sizeof(vect<pos_type> ) * xsize));
		CUDA_CHECK(hipMalloc(&y, sizeof(std::pair<part_iter, part_iter>) * ysize));
		CUDA_CHECK(hipMalloc(&z, sizeof(multi_src) * zsize));
		CUDA_CHECK(hipMalloc(&xi, sizeof(int) * isize));
		CUDA_CHECK(hipMalloc(&yi, sizeof(int) * isize));
		CUDA_CHECK(hipMalloc(&zi, sizeof(int) * isize));
		CUDA_CHECK(hipHostMalloc(&fp, sizeof(force) * xsize));
		CUDA_CHECK(hipHostMalloc(&xp, sizeof(vect<pos_type> ) * xsize));
		CUDA_CHECK(hipHostMalloc(&yp, sizeof(std::pair<part_iter, part_iter>) * ysize));
		CUDA_CHECK(hipHostMalloc(&zp, sizeof(multi_src) * zsize));
		CUDA_CHECK(hipHostMalloc(&xip, sizeof(int) * isize));
		CUDA_CHECK(hipHostMalloc(&yip, sizeof(int) * isize));
		CUDA_CHECK(hipHostMalloc(&zip, sizeof(int) * isize));
		CUDA_CHECK(hipStreamCreate(&stream));
	}
	void resize(int xs, int ys, int zs, int is) {
		if (xs > xsize) {
			while (xsize < xs) {
				xsize *= 2;
			}
			CUDA_CHECK(hipFree(x));
			CUDA_CHECK(hipFree(f));
			CUDA_CHECK(hipMalloc(&f, sizeof(force) * xsize));
			CUDA_CHECK(hipMalloc(&x, sizeof(vect<pos_type> ) * xsize));
			CUDA_CHECK(hipHostFree(xp));
			CUDA_CHECK(hipHostFree(fp));
			CUDA_CHECK(hipHostMalloc(&fp, sizeof(force) * xsize));
			CUDA_CHECK(hipHostMalloc(&xp, sizeof(vect<pos_type> ) * xsize));
		}
		if (ys > ysize) {
			while (ysize < ys) {
				ysize *= 2;
			}
			CUDA_CHECK(hipFree(y));
			CUDA_CHECK(hipMalloc(&y, sizeof(std::pair<part_iter, part_iter>) * ysize));
			CUDA_CHECK(hipHostFree(yp));
			CUDA_CHECK(hipHostMalloc(&yp, sizeof(std::pair<part_iter, part_iter>) * ysize));
		}
		if (zs > zsize) {
			while (zsize < zs) {
				zsize *= 2;
			}
			CUDA_CHECK(hipFree(z));
			CUDA_CHECK(hipMalloc(&z, sizeof(multi_src) * zsize));
			CUDA_CHECK(hipHostFree(zp));
			CUDA_CHECK(hipHostMalloc(&zp, sizeof(multi_src) * zsize));
		}
		if (is > isize) {
			while (isize < is) {
				isize *= 2;
			}
			CUDA_CHECK(hipFree(xi));
			CUDA_CHECK(hipFree(yi));
			CUDA_CHECK(hipFree(zi));
			CUDA_CHECK(hipMalloc(&xi, sizeof(int) * isize));
			CUDA_CHECK(hipMalloc(&yi, sizeof(int) * isize));
			CUDA_CHECK(hipMalloc(&zi, sizeof(int) * isize));
			CUDA_CHECK(hipHostFree(xip));
			CUDA_CHECK(hipHostFree(yip));
			CUDA_CHECK(hipHostFree(zip));
			CUDA_CHECK(hipHostMalloc(&xip, sizeof(int) * isize));
			CUDA_CHECK(hipHostMalloc(&yip, sizeof(int) * isize));
			CUDA_CHECK(hipHostMalloc(&zip, sizeof(int) * isize));
		}
	}
};

static std::atomic<int> lock(0);
static std::stack<cuda_context> stack;

cuda_context pop_context(int xs, int ys, int zs, int is) {
	while (lock++ != 0) {
		lock--;
	}
	if (stack.empty()) {
		lock--;
		return cuda_context(xs, ys, zs, is);
	} else {
		auto ctx = stack.top();
		stack.pop();
		lock--;
		ctx.resize(xs, ys, zs, is);
		return ctx;
	}
}

void push_context(cuda_context ctx) {
	while (lock++ != 0) {
		lock--;
	}
	stack.push(ctx);
	lock--;
}

std::uint64_t gravity_PP_direct_cuda(std::vector<cuda_work_unit> &&units) {
	thread_cnt++;

	static const auto opts = options::get();
	static const float m = opts.m_tot / opts.problem_size;
	static thread_local std::vector<int> xindex;
	static thread_local std::vector<int> yindex;
	static thread_local std::vector<int> zindex;
	static thread_local std::vector<force> f;
	static thread_local std::vector<vect<pos_type>> x;
	static thread_local std::vector<std::pair<part_iter, part_iter>> y;
	static thread_local std::vector<multi_src> z;
	xindex.resize(0);
	yindex.resize(0);
	zindex.resize(0);
	f.resize(0);
	x.resize(0);
	y.resize(0);
	z.resize(0);

	int xi = 0;
	int yi = 0;
	int zi = 0;
	std::uint64_t interactions = 0;
	for (const auto &unit : units) {
		xindex.push_back(xi);
		yindex.push_back(yi);
		zindex.push_back(zi);
		xi += unit.xptr->size();
		yi += unit.yiters.size();
		zi += unit.z.size();
		f.insert(f.end(), unit.fptr->begin(), unit.fptr->end());
		x.insert(x.end(), unit.xptr->begin(), unit.xptr->end());
		for (int j = 0; j < unit.yiters.size(); j++) {
			std::pair<part_iter, part_iter> iter = unit.yiters[j];
			iter.first -= y_begin;
			iter.second -= y_begin;
			interactions += unit.xptr->size() * (iter.second - iter.first);
			y.push_back(iter);
		}
		for (int j = 0; j < unit.z.size(); j++) {
			z.push_back(*unit.z[j]);
		}
	}
	xindex.push_back(xi);
	yindex.push_back(yi);
	zindex.push_back(zi);
	const auto fbytes = sizeof(force) * f.size();
	const auto xbytes = sizeof(vect<pos_type> ) * x.size();
	const auto ybytes = sizeof(std::pair<part_iter, part_iter>) * y.size();
	const auto zbytes = sizeof(multi_src) * z.size();
	const auto xibytes = sizeof(int) * xindex.size();
	const auto yibytes = sizeof(int) * yindex.size();
	const auto zibytes = sizeof(int) * zindex.size();

	auto ctx = pop_context(x.size(), y.size(), z.size(), zindex.size());
	memcpy(ctx.fp, f.data(), fbytes);
	memcpy(ctx.xp, x.data(), xbytes);
	memcpy(ctx.yp, y.data(), ybytes);
	memcpy(ctx.zp, z.data(), zbytes);
	memcpy(ctx.xip, xindex.data(), xibytes);
	memcpy(ctx.yip, yindex.data(), yibytes);
	memcpy(ctx.zip, zindex.data(), zibytes);
	CUDA_CHECK(hipMemcpyAsync(ctx.f, ctx.fp, fbytes, hipMemcpyHostToDevice, ctx.stream));
	CUDA_CHECK(hipMemcpyAsync(ctx.y, ctx.yp, ybytes, hipMemcpyHostToDevice, ctx.stream));
	if (zbytes != 0) {
//		printf( "%li %lli %lli\n", zbytes, ctx.z, ctx.zp);
		CUDA_CHECK(hipMemcpyAsync(ctx.z, ctx.zp, zbytes, hipMemcpyHostToDevice, ctx.stream));
	}
	CUDA_CHECK(hipMemcpyAsync(ctx.x, ctx.xp, xbytes, hipMemcpyHostToDevice, ctx.stream));
	CUDA_CHECK(hipMemcpyAsync(ctx.yi, ctx.yip, yibytes, hipMemcpyHostToDevice, ctx.stream));
	CUDA_CHECK(hipMemcpyAsync(ctx.xi, ctx.xip, xibytes, hipMemcpyHostToDevice, ctx.stream));
	CUDA_CHECK(hipMemcpyAsync(ctx.zi, ctx.zip, zibytes, hipMemcpyHostToDevice, ctx.stream));

PPPC_direct_kernel<<<dim3(units.size(),1,1),dim3(WARPSIZE,NWARP,1),0,ctx.stream>>>(ctx.f,ctx.x,y_vect, ctx.y,ctx.z,ctx.xi,ctx.yi,ctx.zi, m, opts.soft_len, opts.ewald);

										CUDA_CHECK(hipMemcpyAsync(ctx.fp, ctx.f, fbytes, hipMemcpyDeviceToHost, ctx.stream));
	while (hipStreamQuery(ctx.stream) != hipSuccess) {
		yield_to_hpx();
	}
	int k = 0;
	for (const auto &unit : units) {
		for (auto &this_f : *unit.fptr) {
			this_f = ctx.fp[k];
			k++;
		}
	}
	push_context(ctx);
	thread_cnt--;
	return interactions * 36;
}

